#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <list>
#include <vector>
#include <sstream>
#include <stdexcept>
#include <curl/curl.h>
#include <hip/hip_runtime.h>

/**
 * @brief CUDA kernel to predict the spot price of energy based on input features.
 *
 * @param d_demand Array of scheduled demand values.
 * @param d_generation Array of scheduled generation values.
 * @param d_semi_scheduled Array of semi-scheduled generation values.
 * @param d_net_import Array of net import values.
 * @param d_weather_data Array of weather data values.
 * @param d_spot_price Output array for predicted spot prices.
 * @param n Size of the input arrays.
 */
__global__ void predictSpotPriceKernel(double* d_demand, double* d_generation, double* d_semi_scheduled, double* d_net_import, double* d_weather_data, double* d_spot_price, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        // Basic linear combination model for prediction (this should be replaced with a proper model)
        d_spot_price[idx] = 0.5 * d_demand[idx] + 0.2 * d_generation[idx] + 0.1 * d_semi_scheduled[idx] + 0.1 * d_net_import[idx] + 0.1 * d_weather_data[idx];
    }
}

/**
 * @brief Function to predict the spot prices using CUDA.
 *
 * @param demand Vector of scheduled demand values.
 * @param generation Vector of scheduled generation values.
 * @param semi_scheduled Vector of semi-scheduled generation values.
 * @param net_import Vector of net import values.
 * @param weather_data Vector of weather data values.
 * @return Vector of predicted spot prices.
 */
std::vector<double> predictSpotPrice(const std::vector<double>& demand, const std::vector<double>& generation, const std::vector<double>& semi_scheduled, const std::vector<double>& net_import, const std::vector<double>& weather_data) {
    int n = demand.size();
    std::vector<double> spot_price(n);

    double* d_demand, * d_generation, * d_semi_scheduled, * d_net_import, * d_weather_data, * d_spot_price;

    // Allocate device memory
    hipMalloc(&d_demand, n * sizeof(double));
    hipMalloc(&d_generation, n * sizeof(double));
    hipMalloc(&d_semi_scheduled, n * sizeof(double));
    hipMalloc(&d_net_import, n * sizeof(double));
    hipMalloc(&d_weather_data, n * sizeof(double));
    hipMalloc(&d_spot_price, n * sizeof(double));

    // Copy data from host to device
    hipMemcpy(d_demand, demand.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_generation, generation.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_semi_scheduled, semi_scheduled.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_net_import, net_import.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_weather_data, weather_data.data(), n * sizeof(double), hipMemcpyHostToDevice);

    // Launch the kernel with 256 threads per block
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    predictSpotPriceKernel << <numBlocks, blockSize >> > (d_demand, d_generation, d_semi_scheduled, d_net_import, d_weather_data, d_spot_price, n);

    // Copy the result back to host
    hipMemcpy(spot_price.data(), d_spot_price, n * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_demand);
    hipFree(d_generation);
    hipFree(d_semi_scheduled);
    hipFree(d_net_import);
    hipFree(d_weather_data);
    hipFree(d_spot_price);

    return spot_price;
}

/**
 * @brief CUDA kernel to scale features using min-max normalization.
 *
 * @param d_data Input array to be scaled.
 * @param d_scaled_data Output array for scaled values.
 * @param min Minimum value in the dataset.
 * @param max Maximum value in the dataset.
 * @param n Size of the input array.
 */
__global__ void scaleFeaturesKernel(double* d_data, double* d_scaled_data, double min, double max, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_scaled_data[idx] = (d_data[idx] - min) / (max - min);
    }
}

/**
 * @brief Function to scale features using min-max normalization.
 *
 * @param data Vector of input values to be scaled.
 * @return Vector of scaled values.
 */
std::vector<double> scaleFeatures(const std::vector<double>& data) {
    int n = data.size();
    std::vector<double> scaled_data(n);

    double* d_data, * d_scaled_data;

    // Find min and max in the data
    double min_val = *std::min_element(data.begin(), data.end());
    double max_val = *std::max_element(data.begin(), data.end());

    // Allocate device memory
    hipMalloc(&d_data, n * sizeof(double));
    hipMalloc(&d_scaled_data, n * sizeof(double));

    // Copy data from host to device
    hipMemcpy(d_data, data.data(), n * sizeof(double), hipMemcpyHostToDevice);

    // Launch the kernel
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    scaleFeaturesKernel << <numBlocks, blockSize >> > (d_data, d_scaled_data, min_val, max_val, n);

    // Copy the result back to host
    hipMemcpy(scaled_data.data(), d_scaled_data, n * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_data);
    hipFree(d_scaled_data);

    return scaled_data;
}

/**
 * @brief CUDA kernel for predicting spot price using a polynomial regression model.
 *
 * @param d_demand Array of scaled demand values.
 * @param d_generation Array of scaled generation values.
 * @param d_semi_scheduled Array of scaled semi-scheduled generation values.
 * @param d_net_import Array of scaled net import values.
 * @param d_weather_data Array of scaled weather data values.
 * @param d_spot_price Output array for predicted spot prices.
 * @param n Size of the input arrays.
 */
__global__ void predictSpotPricePolyKernel(double* d_demand, double* d_generation, double* d_semi_scheduled, double* d_net_import, double* d_weather_data, double* d_spot_price, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        // Polynomial regression model for prediction
        d_spot_price[idx] = 0.3 * d_demand[idx] * d_demand[idx]
            + 0.4 * d_generation[idx] * d_generation[idx]
            + 0.1 * d_semi_scheduled[idx] * d_semi_scheduled[idx]
            + 0.1 * d_net_import[idx]
            + 0.1 * d_weather_data[idx];
    }
}

/**
 * @brief Function to predict the spot prices using a polynomial regression model with CUDA.
 *
 * @param demand Vector of scaled demand values.
 * @param generation Vector of scaled generation values.
 * @param semi_scheduled Vector of scaled semi-scheduled generation values.
 * @param net_import Vector of scaled net import values.
 * @param weather_data Vector of scaled weather data values.
 * @return Vector of predicted spot prices.
 */
std::vector<double> predictSpotPricePoly(const std::vector<double>& demand, const std::vector<double>& generation, const std::vector<double>& semi_scheduled, const std::vector<double>& net_import, const std::vector<double>& weather_data) {
    int n = demand.size();
    std::vector<double> spot_price(n);

    double* d_demand, * d_generation, * d_semi_scheduled, * d_net_import, * d_weather_data, * d_spot_price;

    // Allocate device memory
    hipMalloc(&d_demand, n * sizeof(double));
    hipMalloc(&d_generation, n * sizeof(double));
    hipMalloc(&d_semi_scheduled, n * sizeof(double));
    hipMalloc(&d_net_import, n * sizeof(double));
    hipMalloc(&d_weather_data, n * sizeof(double));
    hipMalloc(&d_spot_price, n * sizeof(double));

    // Copy data from host to device
    hipMemcpy(d_demand, demand.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_generation, generation.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_semi_scheduled, semi_scheduled.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_net_import, net_import.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_weather_data, weather_data.data(), n * sizeof(double), hipMemcpyHostToDevice);

    // Launch the kernel
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    predictSpotPricePolyKernel << <numBlocks, blockSize >> > (d_demand, d_generation, d_semi_scheduled, d_net_import, d_weather_data, d_spot_price, n);

    // Copy the result back to host
    hipMemcpy(spot_price.data(), d_spot_price, n * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_demand);
    hipFree(d_generation);
    hipFree(d_semi_scheduled);
    hipFree(d_net_import);
    hipFree(d_weather_data);
    hipFree(d_spot_price);

    return spot_price;
}

/**
 * @brief CUDA kernel to calculate Mean Absolute Error (MAE) between predicted and actual spot prices.
 *
 * @param d_actual Array of actual spot prices.
 * @param d_predicted Array of predicted spot prices.
 * @param d_errors Output array for absolute errors.
 * @param n Size of the input arrays.
 */
__global__ void calculateMAEKernel(double* d_actual, double* d_predicted, double* d_errors, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_errors[idx] = abs(d_actual[idx] - d_predicted[idx]);
    }
}

/**
 * @brief Function to calculate Mean Absolute Error (MAE) between predicted and actual spot prices using CUDA.
 *
 * @param actual Vector of actual spot prices.
 * @param predicted Vector of predicted spot prices.
 * @return The Mean Absolute Error (MAE).
 */
double calculateMAE(const std::vector<double>& actual, const std::vector<double>& predicted) {
    int n = actual.size();
    std::vector<double> errors(n);

    double* d_actual, * d_predicted, * d_errors;

    // Allocate device memory
    hipMalloc(&d_actual, n * sizeof(double));
    hipMalloc(&d_predicted, n * sizeof(double));
    hipMalloc(&d_errors, n * sizeof(double));

    // Copy data from host to device
    hipMemcpy(d_actual, actual.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_predicted, predicted.data(), n * sizeof(double), hipMemcpyHostToDevice);

    // Launch the kernel
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    calculateMAEKernel << <numBlocks, blockSize >> > (d_actual, d_predicted, d_errors, n);

    // Copy the result back to host
    hipMemcpy(errors.data(), d_errors, n * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_actual);
    hipFree(d_predicted);
    hipFree(d_errors);

    // Calculate the mean of errors
    double mae = std::accumulate(errors.begin(), errors.end(), 0.0) / n;

    return mae;
}

std::vector<std::string> splitString(const std::string& str, char delimiter) {
    std::vector<std::string> result;
    std::stringstream ss(str);
    std::string item;

    while (std::getline(ss, item, delimiter)) {
        result.push_back(item);
    }

    return result;
}

class AEMO_data
{
public:
    AEMO_data(std::string date, double Spot_Price, double Scheduled_Demand, double Scheduled_Generation, double Semi_Scheduled_Generation, double net_import, std::string type);
    ~AEMO_data();

    static AEMO_data format_data(const std::string& raw_data);
};

AEMO_data::AEMO_data(std::string date, double Spot_Price, double Scheduled_Demand, double Scheduled_Generation, double Semi_Scheduled_Generation, double net_import, std::string type)
{
}

AEMO_data::~AEMO_data()
{
}

AEMO_data AEMO_data::format_data(const std::string& raw_data) {
    auto split = splitString(raw_data, ',');

    if (split.size() != 7)
        throw std::invalid_argument("size of string must be 7");

    if (split[0].empty() || split[1].empty() || split[2].empty() || split[3].empty() || split[4].empty() || split[5].empty() || split[6].empty())
        throw std::invalid_argument("Split data had null elements");

    return AEMO_data(split[0], std::stod(split[1]), std::stod(split[2]), std::stod(split[3]), std::stod(split[4]), std::stod(split[5]), split[6]);
}

// This function will be called by libcurl to write received data
size_t WriteCallback(void* contents, size_t size, size_t nmemb, std::string* s) {
    size_t totalSize = size * nmemb;
    s->append((char*)contents, totalSize);
    return totalSize;
}

void getWeatherData() {
    CURL* curl;
    CURLcode res;
    std::string readBuffer;

    curl = curl_easy_init();  // Initialize a CURL session
    if (curl) {
        // Set the URL for the request
        curl_easy_setopt(curl, CURLOPT_URL, "http://www.example.com");

        // Set the write function callback to store the response data
        curl_easy_setopt(curl, CURLOPT_WRITEFUNCTION, WriteCallback);

        // Set the pointer to the response data
        curl_easy_setopt(curl, CURLOPT_WRITEDATA, &readBuffer);

        // Perform the request, res will get the return code
        res = curl_easy_perform(curl);

        // Check for errors
        if (res != CURLE_OK)
            std::cerr << "curl_easy_perform() failed: " << curl_easy_strerror(res) << std::endl;
        else
            std::cout << "Response data: " << readBuffer << std::endl;

        // Always cleanup
        curl_easy_cleanup(curl);
    }
}

int main() {
    std::string temp;
    std::ifstream data_file("current_data.csv");
    std::list<AEMO_data> data;

    CURL* curl; 

    while (std::getline(data_file, temp)) {
        if (temp[0] == 'S')
            continue;

        data.push_back(AEMO_data::format_data(temp));
    }

    data_file.close();
}
